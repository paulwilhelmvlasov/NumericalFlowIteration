#include <hip/hip_runtime.h>

__global__
void kernel( float *x, float *y, float *result )
{
    size_t index = blockDim.x*blockIdx.x + threadIdx.x;
    result += index;
    x += index;
    y += index;
    *result = *x + *y;
}


void run_kernel()
{
    float *x { nullptr }, *y { nullptr }, *result { nullptr };
    hipMalloc( &x, 1024*sizeof(float) );
    hipMalloc( &y, 1024*sizeof(float) );
    hipMalloc( &result, 1024*sizeof(float) );

    kernel<<<4,32>>> (x,y,result);

    hipFree( result );
    hipFree( y );
    hipFree( y );


}

